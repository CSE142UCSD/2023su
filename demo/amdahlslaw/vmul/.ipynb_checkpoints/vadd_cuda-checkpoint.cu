#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <unistd.h>
#include <stdint.h>
#include <inttypes.h>
#include "microbench_cuda.h"
#define BLOCK_SIZE 1024


extern "C" 
uint64_t *vector_mul_double_cuda(double *va, double *vb, unsigned long int size, int grids)
{
         double *d_va, *d_vb;
         hipMalloc((void **) &d_va, sizeof(double)*size);
         hipMalloc((void **) &d_vb, sizeof(double)*size);
         hipMemcpy(d_array, _array, sizeof(double)*size, hipMemcpyHostToDevice);

         unsigned int grid_cols = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

         // Launch kernel 
         baseline_double_cuda_init<<<grid_cols, BLOCK_SIZE>>>(d_array, size);
         for(int j=0;j<3;j++)
         {
             baseline_double_cuda_kernel<<<grid_cols, BLOCK_SIZE>>>(d_array, d_output, size, j);
             hipDeviceSynchronize();
             hipMemcpy(d_array, d_output, sizeof(double)*size, hipMemcpyDeviceToDevice);
         }
         // Transefr results from device to host 
         hipMemcpy(_array, d_output, sizeof(double)*size, hipMemcpyDeviceToHost);
         hipFree(d_output);
         hipFree(d_array);
	 return _array;
}
__global__ void baseline_double_cuda_kernel(double *input, double *output, int size, int j) 
{
    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos < size)
    {
        output[pos] = input[pos]+pos/(j+1)+input[pos-1];
    }
}
__global__ void baseline_double_cuda_init(double *array, int size) 
{
    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos < size) 
    {
        array[pos] = 0;
    }
}
